#include "hip/hip_runtime.h"
#include <../src/vec/is/sf/impls/basic/sfpack.h>
#include <hip/hip_runtime.h>

/*====================================================================================*/
/*  Templated CUDA kernels for pack/unpack. The Op can be regular or atomic           */
/*====================================================================================*/

/* Suppose user calls PetscSFReduce(sf,unit,...) and <unit> is an MPI data type made of 16 PetscReals, then
   <Type> is PetscReal, which is the primitive type we operate on.
   <bs>   is 16, which says <unit> contains 16 primitive types.
   <BS>   is 8, which is the maximal SIMD width we will try to vectorize operations on <unit>.
   <EQ>   is 0, which is (bs == BS ? 1 : 0)

  If instead, <unit> has 8 PetscReals, then bs=8, BS=8, EQ=1, rendering MBS below to a compile time constant.
  For the common case in VecScatter, bs=1, BS=1, EQ=1, MBS=1, the inner for-loops below will be totally unrolled.
*/
template<class Type,PetscInt BS,PetscInt EQ>
__global__ static void d_Pack(PetscInt count,const PetscInt *idx,PetscInt bs,const void *unpacked,void *packed)
{
  PetscInt        i,tid = blockIdx.x*blockDim.x + threadIdx.x;
  const PetscInt  grid_size = gridDim.x * blockDim.x;
  const Type      *u = (const Type*)unpacked;
  Type            *p = (Type*)packed;
  const PetscInt  M = (EQ) ? 1 : bs/BS; /* If EQ, then M=1 enables compiler's const-propagation */
  const PetscInt  MBS = M*BS;  /* MBS=bs. We turn MBS into a compile-time const when EQ=1. */

  for (; tid<count; tid += grid_size) {
    if (!idx) {for (i=0; i<MBS; i++) p[tid*MBS+i] = u[tid*MBS+i];}
    else      {for (i=0; i<MBS; i++) p[tid*MBS+i] = u[idx[tid]*MBS+i];}
  }
}

template<class Type,class Op,PetscInt BS,PetscInt EQ>
__global__ static void d_UnpackAndOp(PetscInt count,const PetscInt *idx,PetscInt bs,void *unpacked,const void *packed)
{
  PetscInt        i,tid = blockIdx.x*blockDim.x + threadIdx.x;
  const PetscInt  grid_size = gridDim.x * blockDim.x;
  Type            *u = (Type*)unpacked;
  const Type      *p = (const Type*)packed;
  const PetscInt  M = (EQ) ? 1 : bs/BS, MBS = M*BS;
  Op              op;

  for (; tid<count; tid += grid_size) {
    if (!idx) {for (i=0; i<MBS; i++) op(u[tid*MBS+i],     p[tid*MBS+i]);}
    else      {for (i=0; i<MBS; i++) op(u[idx[tid]*MBS+i],p[tid*MBS+i]);}
  }
}

template<class Type,class Op,PetscInt BS,PetscInt EQ>
__global__ static void d_FetchAndOp(PetscInt count,const PetscInt *idx,PetscInt bs,void *unpacked,void *packed)
{
  PetscInt        i,tid = blockIdx.x*blockDim.x + threadIdx.x;
  const PetscInt  grid_size = gridDim.x * blockDim.x;
  Type            *u = (Type*)unpacked,*p;
  const PetscInt  M = (EQ) ? 1 : bs/BS, MBS = M*BS;
  Op              op;

  for (; tid<count; tid += grid_size) {
    if (!idx) {for (i=0; i<MBS; i++) p[tid*MBS+i] = op(u[tid*MBS+i],p[tid*MBS+i]);}
    else      {for (i=0; i<MBS; i++) p[tid*MBS+i] = op(u[idx[tid]*MBS+i],p[tid*MBS+i]);}
  }
}

/*====================================================================================*/
/*                             Regular operations on device                           */
/*====================================================================================*/
template<typename Type> struct Insert {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = y;             return old;}};
template<typename Type> struct Add    {__device__ Type operator() (Type& x,Type y) const {Type old = x; x += y;             return old;}};
template<typename Type> struct Mult   {__device__ Type operator() (Type& x,Type y) const {Type old = x; x *= y;             return old;}};
template<typename Type> struct Min    {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = PetscMin(x,y); return old;}};
template<typename Type> struct Max    {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = PetscMax(x,y); return old;}};
template<typename Type> struct LAND   {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = x && y;        return old;}};
template<typename Type> struct LOR    {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = x || y;        return old;}};
template<typename Type> struct LXOR   {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = !x != !y;      return old;}};
template<typename Type> struct BAND   {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = x & y;         return old;}};
template<typename Type> struct BOR    {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = x | y;         return old;}};
template<typename Type> struct BXOR   {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = x ^ y;         return old;}};
template<typename Type> struct Minloc {
  __device__ Type operator() (Type& x,Type y) const {
    Type old = x;
    if (y.a < x.a) x = y;
    else if (y.a == x.a) x.b = min(x.b,y.b);
    return old;
  }
};
template<typename Type> struct Maxloc {
  __device__ Type operator() (Type& x,Type y) const {
    Type old = x;
    if (y.a > x.a) x = y;
    else if (y.a == x.a) x.b = min(x.b,y.b); /* See MPI MAXLOC */
    return old;
  }
};

/*====================================================================================*/
/*                             Atomic operations on device                            */
/*====================================================================================*/

/*
  Atomic Insert (exchange) operations

  CUDA C Programming Guide V10.1 Chapter B.12.1.3:

  int atomicExch(int* address, int val);
  unsigned int atomicExch(unsigned int* address, unsigned int val);
  unsigned long long int atomicExch(unsigned long long int* address, unsigned long long int val);
  float atomicExch(float* address, float val);

  reads the 32-bit or 64-bit word old located at the address address in global or shared
  memory and stores val back to memory at the same address. These two operations are
  performed in one atomic transaction. The function returns old.

  PETSc notes:

  It may be useful in PetscSFFetchAndOp with op = MPIU_REPLACE.

  VecScatter with multiple entries scattered to the same location using INSERT_VALUES does not need
  atomic insertion, since it does not need the old value. A 32-bit or 64-bit store instruction should
  be atomic itself.

  With bs>1 and a unit > 64 bits, the current element-wise atomic approach can not guarantee the whole
  insertion is atomic. Hope no user codes rely on that.
*/

#if defined(PETSC_USE_REAL_DOUBLE)
__device__ static double atomicExch(double* address,double val) {return __longlong_as_double(atomicExch((unsigned long long int*)address,__double_as_longlong(val)));}
#endif

#if defined(PETSC_USE_64BIT_INDICES)
__device__ static PetscInt atomicExch(PetscInt* address,PetscInt val) {return (PetscInt)(atomicExch((unsigned long long int*)address,(unsigned long long int)val));}
#endif

template<typename Type> struct AtomicInsert {__device__ Type operator() (Type& x,Type y) const {return atomicExch(&x,y);}};

/*
  Atomic add operations

  CUDA C Programming Guide V10.1 Chapter B.12.1.1:

  int atomicAdd(int* address, int val);
  unsigned int atomicAdd(unsigned int* address,unsigned int val);
  unsigned long long int atomicAdd(unsigned long long int* address,unsigned long long int val);
  float atomicAdd(float* address, float val);
  double atomicAdd(double* address, double val);
  __half2 atomicAdd(__half2 *address, __half2 val);
  __half atomicAdd(__half *address, __half val);

  reads the 16-bit, 32-bit or 64-bit word old located at the address address in global or shared memory, computes (old + val),
  and stores the result back to memory at the same address. These three operations are performed in one atomic transaction. The
  function returns old.

  The 32-bit floating-point version of atomicAdd() is only supported by devices of compute capability 2.x and higher.
  The 64-bit floating-point version of atomicAdd() is only supported by devices of compute capability 6.x and higher.
  The 32-bit __half2 floating-point version of atomicAdd() is only supported by devices of compute capability 6.x and
  higher. The atomicity of the __half2 add operation is guaranteed separately for each of the two __half elements;
  the entire __half2 is not guaranteed to be atomic as a single 32-bit access.
  The 16-bit __half floating-point version of atomicAdd() is only supported by devices of compute capability 7.x and higher.
*/

#if defined(PETSC_USE_64BIT_INDICES)
__device__ static PetscInt atomicAdd(PetscInt* address,PetscInt val) {return (PetscInt)atomicAdd((unsigned long long int*)address,(unsigned long long int)val);}
#endif

template<typename Type> struct AtomicAdd {__device__ Type operator() (Type& x,Type y) const {return atomicAdd(&x,y);}};

template<> struct AtomicAdd<double> {
  __device__ double operator() (double& x,double y) const {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 600)
    return atomicAdd(&x,y);
#else
    double                 *address = &x, val = y;
    unsigned long long int *address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
      assumed = old;
      old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
      /* Note: uses integer comparison to avoid hang in case of NaN (since NaN !=NaN) */
    } while (assumed != old);
    return __longlong_as_double(old);
#endif
  }
};

template<> struct AtomicAdd<float> {
  __device__ float operator() (float& x,float y) const {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 200)
    return atomicAdd(&x,y);
#else
    float *address = &x, val = y;
    int   *address_as_int = (int*)address;
    int   old = *address_as_int, assumed;
    do {
      assumed = old;
      old     = atomicCAS(address_as_int, assumed, __float_as_int(val + __int_as_float(assumed)));
      /* Note: uses integer comparison to avoid hang in case of NaN (since NaN !=NaN) */
    } while (assumed != old);
    return __int_as_float(old);
#endif
  }
};

template<> struct AtomicAdd<PetscComplex> {
 __device__ PetscComplex operator() (PetscComplex& x,PetscComplex y) const {
  PetscComplex         old, *z = &old;
  PetscReal            *xp = (PetscReal*)&x,*yp = (PetscReal*)&y;
  AtomicAdd<PetscReal> op;
  z[0] = op(xp[0],yp[0]);
  z[1] = op(xp[1],yp[1]);
  return old; /* The returned value may not be atomic. It can be mix of two ops. Caller should discard it. */
 }
};

/*
  Atomic Mult operations:

  CUDA has no atomicMult at all, so we build our own with atomicCAS
 */
#if defined(PETSC_USE_REAL_DOUBLE)
__device__ static double atomicMult(double* address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    /* Other threads can access and modify value of *address_as_ull after the read above and before the write below */
    old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(val*__longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#elif defined(PETSC_USE_REAL_SINGLE)
__device__ static float atomicMult(float* address,float val)
{
  int *address_as_int = (int*)(address);
  int old = *address_as_int, assumed;
  do {
    assumed  = old;
    old      = atomicCAS(address_as_int, assumed, __float_as_int(val*__int_as_float(assumed)));
  } while (assumed != old);
  return __int_as_float(old);
}
#endif

__device__ static int atomicMult(int* address,int val)
{
  int *address_as_int = (int*)(address);
  int old = *address_as_int, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_int, assumed, val*assumed);
  } while (assumed != old);
  return (int)old;
}

#if defined(PETSC_USE_64BIT_INDICES)
__device__ static int atomicMult(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(val*(PetscInt)assumed));
  } while (assumed != old);
  return (PetscInt)old;
}
#endif

template<typename Type> struct AtomicMult {__device__ Type operator() (Type& x,Type y) const {return atomicMult(&x,y);}};

/*
  Atomic Min/Max operations

  CUDA C Programming Guide V10.1 Chapter B.12.1.4~5:

  int atomicMin(int* address, int val);
  unsigned int atomicMin(unsigned int* address,unsigned int val);
  unsigned long long int atomicMin(unsigned long long int* address,unsigned long long int val);

  reads the 32-bit or 64-bit word old located at the address address in global or shared
  memory, computes the minimum of old and val, and stores the result back to memory
  at the same address. These three operations are performed in one atomic transaction.
  The function returns old.
  The 64-bit version of atomicMin() is only supported by devices of compute capability 3.5 and higher.

  atomicMax() is similar.
 */

#if defined(PETSC_USE_REAL_DOUBLE)
__device__ static double atomicMin(double* address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(PetscMin(val,__longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}

__device__ static double atomicMax(double* address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed  = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(PetscMax(val,__longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#elif defined(PETSC_USE_REAL_SINGLE)
__device__ static float atomicMin(float* address,float val)
{
  int *address_as_int = (int*)(address);
  int old = *address_as_int, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_int, assumed, __float_as_int(PetscMin(val,__int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

__device__ static float atomicMax(float* address,float val)
{
  int *address_as_int = (int*)(address);
  int old = *address_as_int, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_int, assumed, __float_as_int(PetscMax(val,__int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}
#endif

#if defined(PETSC_USE_64BIT_INDICES)
__device__ static PetscInt atomicMin(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(PetscMin(val,(PetscInt)assumed)));
  } while (assumed != old);
  return (PetscInt)old;
}

__device__ static PetscInt atomicMax(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(PetscMax(val,(PetscInt)assumed)));
  } while (assumed != old);
  return (PetscInt)old;
}
#endif

template<typename Type> struct AtomicMin {__device__ Type operator() (Type& x,Type y) const {return atomicMin(&x,y);}};
template<typename Type> struct AtomicMax {__device__ Type operator() (Type& x,Type y) const {return atomicMax(&x,y);}};

/*
  Atomic bitwise operations

  CUDA C Programming Guide V10.1 Chapter B.12.2.1 ~ B.12.2.3:

  int atomicAnd(int* address, int val);
  unsigned int atomicAnd(unsigned int* address,unsigned int val);
  unsigned long long int atomicAnd(unsigned long long int* address,unsigned long long int val);

  reads the 32-bit or 64-bit word old located at the address address in global or shared
  memory, computes (old & val), and stores the result back to memory at the same
  address. These three operations are performed in one atomic transaction.
  The function returns old.

  The 64-bit version of atomicAnd() is only supported by devices of compute capability 3.5 and higher.

  atomicOr() and atomicXor are similar.
*/

#if defined(PETSC_USE_64BIT_INDICES)
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 350)
__device__ static PetscInt atomicAnd(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(val & (PetscInt)assumed));
  } while (assumed != old);
  return (PetscInt)old;
}
__device__ static PetscInt atomicOr(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(val | (PetscInt)assumed));
  } while (assumed != old);
  return (PetscInt)old;
}

__device__ static PetscInt atomicXor(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(val ^ (PetscInt)assumed));
  } while (assumed != old);
  return (PetscInt)old;
}
#else
__device__ static PetscInt atomicAnd(PetscInt* address,PetscInt val) {return (PetscInt)atomicAnd((unsigned long long int*)address,(unsigned long long int)val);}
__device__ static PetscInt atomicOr (PetscInt* address,PetscInt val) {return (PetscInt)atomicOr ((unsigned long long int*)address,(unsigned long long int)val);}
__device__ static PetscInt atomicXor(PetscInt* address,PetscInt val) {return (PetscInt)atomicXor((unsigned long long int*)address,(unsigned long long int)val);}
#endif
#endif

template<typename Type> struct AtomicBAND {__device__ Type operator() (Type& x,Type y) const {return atomicAnd(&x,y);}};
template<typename Type> struct AtomicBOR  {__device__ Type operator() (Type& x,Type y) const {return atomicOr (&x,y);}};
template<typename Type> struct AtomicBXOR {__device__ Type operator() (Type& x,Type y) const {return atomicXor(&x,y);}};

/*
  Atomic logical operations:

  CUDA has no atomic logical operations at all. We support them on integer types.
*/

/* A template without definition makes any instantiation not using given specializations erroneous at compile time,
   which is what we want since we only support 32-bit and 64-bit integers.
 */
template<typename Type,class Op,int size/* sizeof(Type) */> struct AtomicLogical;

template<typename Type,class Op>
struct AtomicLogical<Type,Op,4> {
  __device__ Type operator()(Type& x,Type y) const {
    int *address_as_int = (int*)(&x);
    int old = *address_as_int, assumed;
    Op op;
    do {
      assumed = old;
      old     = atomicCAS(address_as_int, assumed, (int)(op((Type)assumed,y)));
    } while (assumed != old);
    return (Type)old;
  }
};

template<typename Type,class Op>
struct AtomicLogical<Type,Op,8> {
  __device__ Type operator()(Type& x,Type y) const {
    unsigned long long int *address_as_ull = (unsigned long long int*)(&x);
    unsigned long long int old = *address_as_ull, assumed;
    Op op;
    do {
      assumed = old;
      old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(op((Type)assumed,y)));
    } while (assumed != old);
    return (Type)old;
  }
};

/* Note land/lor/lxor below are different from LAND etc above. Here we pass arguments by value and return result of ops (not old value) */
template<typename Type> struct land {__device__ Type operator()(Type x, Type y) {return x && y;}};
template<typename Type> struct lor  {__device__ Type operator()(Type x, Type y) {return x || y;}};
template<typename Type> struct lxor {__device__ Type operator()(Type x, Type y) {return (!x != !y);}};

template<typename Type> struct AtomicLAND {__device__ Type operator()(Type& x,Type y) const {AtomicLogical<Type,land<Type>,sizeof(Type)> op; return op(x,y);}};
template<typename Type> struct AtomicLOR  {__device__ Type operator()(Type& x,Type y) const {AtomicLogical<Type,lor<Type> ,sizeof(Type)> op; return op(x,y);}};
template<typename Type> struct AtomicLXOR {__device__ Type operator()(Type& x,Type y) const {AtomicLogical<Type,lxor<Type>,sizeof(Type)> op; return op(x,y);}};

/*====================================================================================*/
/*  Wrapper functions on cuda kernels. Function pointers are stored in 'link'         */
/*====================================================================================*/
template<typename Type,PetscInt BS,PetscInt EQ>
static PetscErrorCode Pack(PetscInt count,const PetscInt *idx,PetscSFPack link,PetscSFPackOpt opt,const void *unpacked,void *packed)
{
  hipError_t err;
  PetscInt    nthreads=256;
  PetscInt    nblocks=(count+nthreads-1)/nthreads;

  PetscFunctionBegin;
  if (nblocks > link->MAX_CORESIDENT_THREADS/nthreads) nblocks = link->MAX_CORESIDENT_THREADS/nthreads;
  d_Pack<Type,BS,EQ><<<nblocks,nthreads,0,link->stream>>>(count,idx,link->bs,unpacked,packed);
  err = hipGetLastError();CHKERRCUDA(err);
  PetscFunctionReturn(0);
}

template<typename Type,class Op,PetscInt BS,PetscInt EQ>
static PetscErrorCode UnpackAndOp(PetscInt count,const PetscInt *idx,PetscSFPack link,PetscSFPackOpt opt,void *unpacked,const void *packed)
{
  hipError_t err;
  PetscInt    nthreads=256;
  PetscInt    nblocks=(count+nthreads-1)/nthreads;

  PetscFunctionBegin;
  if (nblocks > link->MAX_CORESIDENT_THREADS/nthreads) nblocks = link->MAX_CORESIDENT_THREADS/nthreads;
  d_UnpackAndOp<Type,Op,BS,EQ><<<nblocks,nthreads,0,link->stream>>>(count,idx,link->bs,unpacked,packed);
  err = hipGetLastError();CHKERRCUDA(err);
  PetscFunctionReturn(0);
}

template<typename Type,class Op,PetscInt BS,PetscInt EQ>
static PetscErrorCode FetchAndOp(PetscInt count,const PetscInt *idx,PetscSFPack link,PetscSFPackOpt opt,void *unpacked,void *packed)
{
  hipError_t err;
  PetscInt    nthreads=256;
  PetscInt    nblocks=(count+nthreads-1)/nthreads;

  PetscFunctionBegin;
  if (nblocks > link->MAX_CORESIDENT_THREADS/nthreads) nblocks = link->MAX_CORESIDENT_THREADS/nthreads;
  d_FetchAndOp<Type,Op,BS,EQ><<<nblocks,nthreads,0,link->stream>>>(count,idx,link->bs,unpacked,packed);
  err = hipGetLastError();CHKERRCUDA(err);
  PetscFunctionReturn(0);
}

/*====================================================================================*/
/*  Init various types and instantiate pack/unpack function pointers                  */
/*====================================================================================*/
template<typename Type,PetscInt BS,PetscInt EQ>
static void PackInit_RealType(PetscSFPack link)
{
  link->d_Pack             = Pack<Type,BS,EQ>;
  link->d_UnpackAndInsert  = UnpackAndOp<Type,Insert<Type>,BS,EQ>;
  link->d_UnpackAndAdd     = UnpackAndOp<Type,Add<Type>   ,BS,EQ>;
  link->d_UnpackAndMult    = UnpackAndOp<Type,Mult<Type>  ,BS,EQ>;
  link->d_UnpackAndMin     = UnpackAndOp<Type,Min<Type>   ,BS,EQ>;
  link->d_UnpackAndMax     = UnpackAndOp<Type,Max<Type>   ,BS,EQ>;

  link->d_FetchAndInsert   = FetchAndOp <Type,Insert<Type>,BS,EQ>;
  link->d_FetchAndAdd      = FetchAndOp <Type,Add<Type>   ,BS,EQ>;
  link->d_FetchAndMult     = FetchAndOp <Type,Mult<Type>  ,BS,EQ>;
  link->d_FetchAndMin      = FetchAndOp <Type,Min<Type>   ,BS,EQ>;
  link->d_FetchAndMax      = FetchAndOp <Type,Max<Type>   ,BS,EQ>;

  /* Pack() is always data race free */
  link->da_UnpackAndInsert = UnpackAndOp<Type,AtomicInsert<Type>,BS,EQ>;
  link->da_UnpackAndAdd    = UnpackAndOp<Type,AtomicAdd<Type>   ,BS,EQ>;
  link->da_UnpackAndMult   = UnpackAndOp<Type,AtomicMult<Type>  ,BS,EQ>;
  link->da_UnpackAndMin    = UnpackAndOp<Type,AtomicMin<Type>   ,BS,EQ>;
  link->da_UnpackAndMax    = UnpackAndOp<Type,AtomicMax<Type>   ,BS,EQ>;

  link->da_FetchAndInsert  = FetchAndOp <Type,AtomicInsert<Type>,BS,EQ>;
  link->da_FetchAndAdd     = FetchAndOp <Type,AtomicAdd<Type>   ,BS,EQ>;
  link->da_FetchAndMult    = FetchAndOp <Type,AtomicMult<Type>  ,BS,EQ>;
  link->da_FetchAndMin     = FetchAndOp <Type,AtomicMin<Type>   ,BS,EQ>;
  link->da_FetchAndMax     = FetchAndOp <Type,AtomicMax<Type>   ,BS,EQ>;
}

/* Have this templated class to specialize for char integers */
template<typename Type,PetscInt BS,PetscInt EQ,PetscInt size/*sizeof(Type)*/>
struct PackInit_IntegerType_Atomic {
  static void Init(PetscSFPack link) {
    link->da_UnpackAndInsert = UnpackAndOp<Type,AtomicInsert<Type>,BS,EQ>;
    link->da_UnpackAndAdd    = UnpackAndOp<Type,AtomicAdd<Type>   ,BS,EQ>;
    link->da_UnpackAndMult   = UnpackAndOp<Type,AtomicMult<Type>  ,BS,EQ>;
    link->da_UnpackAndMin    = UnpackAndOp<Type,AtomicMin<Type>   ,BS,EQ>;
    link->da_UnpackAndMax    = UnpackAndOp<Type,AtomicMax<Type>   ,BS,EQ>;
    link->da_UnpackAndLAND   = UnpackAndOp<Type,AtomicLAND<Type>  ,BS,EQ>;
    link->da_UnpackAndLOR    = UnpackAndOp<Type,AtomicLOR<Type>   ,BS,EQ>;
    link->da_UnpackAndLXOR   = UnpackAndOp<Type,AtomicLXOR<Type>  ,BS,EQ>;
    link->da_UnpackAndBAND   = UnpackAndOp<Type,AtomicBAND<Type>  ,BS,EQ>;
    link->da_UnpackAndBOR    = UnpackAndOp<Type,AtomicBOR<Type>   ,BS,EQ>;
    link->da_UnpackAndBXOR   = UnpackAndOp<Type,AtomicBXOR<Type>  ,BS,EQ>;

    link->da_FetchAndInsert  = FetchAndOp <Type,AtomicInsert<Type>,BS,EQ>;
    link->da_FetchAndAdd     = FetchAndOp <Type,AtomicAdd<Type>   ,BS,EQ>;
    link->da_FetchAndMult    = FetchAndOp <Type,AtomicMult<Type>  ,BS,EQ>;
    link->da_FetchAndMin     = FetchAndOp <Type,AtomicMin<Type>   ,BS,EQ>;
    link->da_FetchAndMax     = FetchAndOp <Type,AtomicMax<Type>   ,BS,EQ>;
    link->da_FetchAndLAND    = FetchAndOp <Type,AtomicLAND<Type>  ,BS,EQ>;
    link->da_FetchAndLOR     = FetchAndOp <Type,AtomicLOR<Type>   ,BS,EQ>;
    link->da_FetchAndLXOR    = FetchAndOp <Type,AtomicLXOR<Type>  ,BS,EQ>;
    link->da_FetchAndBAND    = FetchAndOp <Type,AtomicBAND<Type>  ,BS,EQ>;
    link->da_FetchAndBOR     = FetchAndOp <Type,AtomicBOR<Type>   ,BS,EQ>;
    link->da_FetchAndBXOR    = FetchAndOp <Type,AtomicBXOR<Type>  ,BS,EQ>;
  }
};

/* CUDA does not support atomics on chars. It is TBD in PETSc. */
template<typename Type,PetscInt BS,PetscInt EQ>
struct PackInit_IntegerType_Atomic<Type,BS,EQ,1> {
  static void Init(PetscSFPack link) {/* Nothing to leave function pointers NULL */}
};

template<typename Type,PetscInt BS,PetscInt EQ>
static void PackInit_IntegerType(PetscSFPack link)
{
  link->d_Pack            = Pack<Type,BS,EQ>;
  link->d_UnpackAndInsert = UnpackAndOp<Type,Insert<Type>,BS,EQ>;
  link->d_UnpackAndAdd    = UnpackAndOp<Type,Add<Type>   ,BS,EQ>;
  link->d_UnpackAndMult   = UnpackAndOp<Type,Mult<Type>  ,BS,EQ>;
  link->d_UnpackAndMin    = UnpackAndOp<Type,Min<Type>   ,BS,EQ>;
  link->d_UnpackAndMax    = UnpackAndOp<Type,Max<Type>   ,BS,EQ>;
  link->d_UnpackAndLAND   = UnpackAndOp<Type,LAND<Type>  ,BS,EQ>;
  link->d_UnpackAndLOR    = UnpackAndOp<Type,LOR<Type>   ,BS,EQ>;
  link->d_UnpackAndLXOR   = UnpackAndOp<Type,LXOR<Type>  ,BS,EQ>;
  link->d_UnpackAndBAND   = UnpackAndOp<Type,BAND<Type>  ,BS,EQ>;
  link->d_UnpackAndBOR    = UnpackAndOp<Type,BOR<Type>   ,BS,EQ>;
  link->d_UnpackAndBXOR   = UnpackAndOp<Type,BXOR<Type>  ,BS,EQ>;

  link->d_FetchAndInsert  = FetchAndOp <Type,Insert<Type>,BS,EQ>;
  link->d_FetchAndAdd     = FetchAndOp <Type,Add<Type>   ,BS,EQ>;
  link->d_FetchAndMult    = FetchAndOp <Type,Mult<Type>  ,BS,EQ>;
  link->d_FetchAndMin     = FetchAndOp <Type,Min<Type>   ,BS,EQ>;
  link->d_FetchAndMax     = FetchAndOp <Type,Max<Type>   ,BS,EQ>;
  link->d_FetchAndLAND    = FetchAndOp <Type,LAND<Type>  ,BS,EQ>;
  link->d_FetchAndLOR     = FetchAndOp <Type,LOR<Type>   ,BS,EQ>;
  link->d_FetchAndLXOR    = FetchAndOp <Type,LXOR<Type>  ,BS,EQ>;
  link->d_FetchAndBAND    = FetchAndOp <Type,BAND<Type>  ,BS,EQ>;
  link->d_FetchAndBOR     = FetchAndOp <Type,BOR<Type>   ,BS,EQ>;
  link->d_FetchAndBXOR    = FetchAndOp <Type,BXOR<Type>  ,BS,EQ>;

  PackInit_IntegerType_Atomic<Type,BS,EQ,sizeof(Type)>::Init(link);
}

#if defined(PETSC_HAVE_COMPLEX)
template<typename Type,PetscInt BS,PetscInt EQ>
static void PackInit_ComplexType(PetscSFPack link)
{
  link->d_Pack            = Pack<Type,BS,EQ>;

  link->d_UnpackAndInsert = UnpackAndOp<Type,Insert<Type>,BS,EQ>;
  link->d_UnpackAndAdd    = UnpackAndOp<Type,Add<Type>   ,BS,EQ>;
  link->d_UnpackAndMult   = UnpackAndOp<Type,Mult<Type>  ,BS,EQ>;
  link->d_FetchAndInsert  = FetchAndOp <Type,Insert<Type>,BS,EQ>;
  link->d_FetchAndAdd     = FetchAndOp <Type,Add<Type>   ,BS,EQ>;
  link->d_FetchAndMult    = FetchAndOp <Type,Mult<Type>  ,BS,EQ>;

  link->da_UnpackAndAdd   = UnpackAndOp<Type,AtomicAdd<Type>,BS,EQ>;
  link->da_UnpackAndMult  = NULL; /* Not implemented yet */
  link->da_FetchAndAdd    = NULL; /* Return value of atomicAdd on complex is not atomic */
}
#endif

typedef signed char                      SignedChar;
typedef unsigned char                    UnsignedChar;
typedef struct {int a;      int b;     } PairInt;
typedef struct {PetscInt a; PetscInt b;} PairPetscInt;

template<typename Type>
static void PackInit_PairType(PetscSFPack link)
{
  link->d_Pack            = Pack<Type,1,1>;
  link->d_UnpackAndInsert = UnpackAndOp<Type,Insert<Type>,1,1>;
  link->d_UnpackAndMinloc = UnpackAndOp<Type,Minloc<Type>,1,1>;
  link->d_UnpackAndMinloc = UnpackAndOp<Type,Minloc<Type>,1,1>;
  link->d_FetchAndInsert  = FetchAndOp <Type,Insert<Type>,1,1>;
  link->d_FetchAndMinloc  = FetchAndOp <Type,Minloc<Type>,1,1>;
  link->d_FetchAndMinloc  = FetchAndOp <Type,Minloc<Type>,1,1>;

  /* Atomics for pair types are not implemented yet */
}

template<typename Type,PetscInt BS,PetscInt EQ>
static void PackInit_DumbType(PetscSFPack link)
{
  link->d_Pack            = Pack<Type,BS,EQ>;
  link->d_UnpackAndInsert = UnpackAndOp<Type,Insert<Type>,BS,EQ>;
  link->d_FetchAndInsert  = FetchAndOp <Type,Insert<Type>,BS,EQ>;

  /* Atomics for dumb types are not implemented yet */
}

/*====================================================================================*/
/*                Main driver to init MPI datatype on device                          */
/*====================================================================================*/

/* Some fields of link are initialized by PetscSFPackSetUp_Host. This routine only does what needed on device */
PetscErrorCode PetscSFPackSetUp_Device(PetscSF sf,PetscSFPack link,MPI_Datatype unit)
{
  PetscErrorCode ierr;
  hipError_t    err;
  PetscInt       nSignedChar=0,nUnsignedChar=0,nInt=0,nPetscInt=0,nPetscReal=0;
  PetscBool      is2Int,is2PetscInt;
#if defined(PETSC_HAVE_COMPLEX)
  PetscInt       nPetscComplex=0;
#endif

  PetscFunctionBegin;
  if (link->deviceinited) PetscFunctionReturn(0);
  ierr = MPIPetsc_Type_compare_contig(unit,MPI_SIGNED_CHAR,  &nSignedChar);CHKERRQ(ierr);
  ierr = MPIPetsc_Type_compare_contig(unit,MPI_UNSIGNED_CHAR,&nUnsignedChar);CHKERRQ(ierr);
  /* MPI_CHAR is treated below as a dumb type that does not support reduction according to MPI standard */
  ierr = MPIPetsc_Type_compare_contig(unit,MPI_INT,  &nInt);CHKERRQ(ierr);
  ierr = MPIPetsc_Type_compare_contig(unit,MPIU_INT, &nPetscInt);CHKERRQ(ierr);
  ierr = MPIPetsc_Type_compare_contig(unit,MPIU_REAL,&nPetscReal);CHKERRQ(ierr);
#if defined(PETSC_HAVE_COMPLEX)
  ierr = MPIPetsc_Type_compare_contig(unit,MPIU_COMPLEX,&nPetscComplex);CHKERRQ(ierr);
#endif
  ierr = MPIPetsc_Type_compare(unit,MPI_2INT,&is2Int);CHKERRQ(ierr);
  ierr = MPIPetsc_Type_compare(unit,MPIU_2INT,&is2PetscInt);CHKERRQ(ierr);

  if (is2Int) {
    PackInit_PairType<PairInt>(link);
  } else if (is2PetscInt) { /* TODO: when is2PetscInt and nPetscInt=2, we don't know which path to take. The two paths support different ops. */
    PackInit_PairType<PairPetscInt>(link);
  } else if (nPetscReal) {
    if      (nPetscReal == 8) PackInit_RealType<PetscReal,8,1>(link); else if (nPetscReal%8 == 0) PackInit_RealType<PetscReal,8,0>(link);
    else if (nPetscReal == 4) PackInit_RealType<PetscReal,4,1>(link); else if (nPetscReal%4 == 0) PackInit_RealType<PetscReal,4,0>(link);
    else if (nPetscReal == 2) PackInit_RealType<PetscReal,2,1>(link); else if (nPetscReal%2 == 0) PackInit_RealType<PetscReal,2,0>(link);
    else if (nPetscReal == 1) PackInit_RealType<PetscReal,1,1>(link); else if (nPetscReal%1 == 0) PackInit_RealType<PetscReal,1,0>(link);
  } else if (nPetscInt) {
    if      (nPetscInt == 8) PackInit_IntegerType<PetscInt,8,1>(link); else if (nPetscInt%8 == 0) PackInit_IntegerType<PetscInt,8,0>(link);
    else if (nPetscInt == 4) PackInit_IntegerType<PetscInt,4,1>(link); else if (nPetscInt%4 == 0) PackInit_IntegerType<PetscInt,4,0>(link);
    else if (nPetscInt == 2) PackInit_IntegerType<PetscInt,2,1>(link); else if (nPetscInt%2 == 0) PackInit_IntegerType<PetscInt,2,0>(link);
    else if (nPetscInt == 1) PackInit_IntegerType<PetscInt,1,1>(link); else if (nPetscInt%1 == 0) PackInit_IntegerType<PetscInt,1,0>(link);
#if defined(PETSC_USE_64BIT_INDICES)
  } else if (nInt) {
    if      (nInt == 8) PackInit_IntegerType<int,8,1>(link); else if (nInt%8 == 0) PackInit_IntegerType<int,8,0>(link);
    else if (nInt == 4) PackInit_IntegerType<int,4,1>(link); else if (nInt%4 == 0) PackInit_IntegerType<int,4,0>(link);
    else if (nInt == 2) PackInit_IntegerType<int,2,1>(link); else if (nInt%2 == 0) PackInit_IntegerType<int,2,0>(link);
    else if (nInt == 1) PackInit_IntegerType<int,1,1>(link); else if (nInt%1 == 0) PackInit_IntegerType<int,1,0>(link);
#endif
  } else if (nSignedChar) {
    if      (nSignedChar == 8) PackInit_IntegerType<SignedChar,8,1>(link); else if (nSignedChar%8 == 0) PackInit_IntegerType<SignedChar,8,0>(link);
    else if (nSignedChar == 4) PackInit_IntegerType<SignedChar,4,1>(link); else if (nSignedChar%4 == 0) PackInit_IntegerType<SignedChar,4,0>(link);
    else if (nSignedChar == 2) PackInit_IntegerType<SignedChar,2,1>(link); else if (nSignedChar%2 == 0) PackInit_IntegerType<SignedChar,2,0>(link);
    else if (nSignedChar == 1) PackInit_IntegerType<SignedChar,1,1>(link); else if (nSignedChar%1 == 0) PackInit_IntegerType<SignedChar,1,0>(link);
  }  else if (nUnsignedChar) {
    if      (nUnsignedChar == 8) PackInit_IntegerType<UnsignedChar,8,1>(link); else if (nUnsignedChar%8 == 0) PackInit_IntegerType<UnsignedChar,8,0>(link);
    else if (nUnsignedChar == 4) PackInit_IntegerType<UnsignedChar,4,1>(link); else if (nUnsignedChar%4 == 0) PackInit_IntegerType<UnsignedChar,4,0>(link);
    else if (nUnsignedChar == 2) PackInit_IntegerType<UnsignedChar,2,1>(link); else if (nUnsignedChar%2 == 0) PackInit_IntegerType<UnsignedChar,2,0>(link);
    else if (nUnsignedChar == 1) PackInit_IntegerType<UnsignedChar,1,1>(link); else if (nUnsignedChar%1 == 0) PackInit_IntegerType<UnsignedChar,1,0>(link);
#if defined(PETSC_HAVE_COMPLEX)
  } else if (nPetscComplex) {
    if      (nPetscComplex == 8) PackInit_ComplexType<PetscComplex,8,1>(link); else if (nPetscComplex%8 == 0) PackInit_ComplexType<PetscComplex,8,0>(link);
    else if (nPetscComplex == 4) PackInit_ComplexType<PetscComplex,4,1>(link); else if (nPetscComplex%4 == 0) PackInit_ComplexType<PetscComplex,4,0>(link);
    else if (nPetscComplex == 2) PackInit_ComplexType<PetscComplex,2,1>(link); else if (nPetscComplex%2 == 0) PackInit_ComplexType<PetscComplex,2,0>(link);
    else if (nPetscComplex == 1) PackInit_ComplexType<PetscComplex,1,1>(link); else if (nPetscComplex%1 == 0) PackInit_ComplexType<PetscComplex,1,0>(link);
#endif
  } else {
    MPI_Aint lb,nbyte;
    ierr = MPI_Type_get_extent(unit,&lb,&nbyte);CHKERRQ(ierr);
    if (lb != 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"Datatype with nonzero lower bound %ld\n",(long)lb);
    if (nbyte % sizeof(int)) { /* If the type size is not multiple of int */
      if      (nbyte == 4) PackInit_DumbType<char,4,1>(link); else if (nbyte%4 == 0) PackInit_DumbType<char,4,0>(link);
      else if (nbyte == 2) PackInit_DumbType<char,2,1>(link); else if (nbyte%2 == 0) PackInit_DumbType<char,2,0>(link);
      else if (nbyte == 1) PackInit_DumbType<char,1,1>(link); else if (nbyte%1 == 0) PackInit_DumbType<char,1,0>(link);
    } else {
      nInt = nbyte / sizeof(int);
      if      (nInt == 8) PackInit_DumbType<int,8,1>(link); else if (nInt%8 == 0) PackInit_DumbType<int,8,0>(link);
      else if (nInt == 4) PackInit_DumbType<int,4,1>(link); else if (nInt%4 == 0) PackInit_DumbType<int,4,0>(link);
      else if (nInt == 2) PackInit_DumbType<int,2,1>(link); else if (nInt%2 == 0) PackInit_DumbType<int,2,0>(link);
      else if (nInt == 1) PackInit_DumbType<int,1,1>(link); else if (nInt%1 == 0) PackInit_DumbType<int,1,0>(link);
    }
  }

  if (!sf_use_default_cuda_stream) {err = hipStreamCreate(&link->stream);CHKERRCUDA(err);}
  if (!sf->MAX_CORESIDENT_THREADS) {
    int                   device;
    struct hipDeviceProp_t props;
    err = hipGetDevice(&device);CHKERRCUDA(err);
    err = hipGetDeviceProperties(&props,device);CHKERRCUDA(err);
    sf->MAX_CORESIDENT_THREADS = props.maxThreadsPerMultiProcessor;
  }
  link->MAX_CORESIDENT_THREADS = sf->MAX_CORESIDENT_THREADS;

  link->deviceinited = PETSC_TRUE;
  PetscFunctionReturn(0);
}
