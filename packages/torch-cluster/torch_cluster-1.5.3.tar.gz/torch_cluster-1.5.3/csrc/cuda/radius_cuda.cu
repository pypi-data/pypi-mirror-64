#include "hip/hip_runtime.h"
#include "radius_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"

#define THREADS 1024

template <typename scalar_t>
__global__ void radius_kernel(const scalar_t *x, const scalar_t *y,
                              const int64_t *ptr_x, const int64_t *ptr_y,
                              int64_t *row, int64_t *col, scalar_t radius,
                              int64_t max_num_neighbors, int64_t dim) {

  const int64_t batch_idx = blockIdx.x;

  const int64_t x_start_idx = ptr_x[batch_idx];
  const int64_t x_end_idx = ptr_x[batch_idx + 1];

  const int64_t y_start_idx = ptr_y[batch_idx];
  const int64_t y_end_idx = ptr_y[batch_idx + 1];

  for (int64_t n_y = y_start_idx + threadIdx.x; n_y < y_end_idx;
       n_y += THREADS) {
    int64_t count = 0;
    for (int64_t n_x = x_start_idx; n_x < x_end_idx; n_x++) {
      scalar_t dist = 0;
      for (int64_t d = 0; d < dim; d++) {
        dist += (x[n_x * dim + d] - y[n_y * dim + d]) *
                (x[n_x * dim + d] - y[n_y * dim + d]);
      }
      dist = sqrt(dist);

      if (dist <= radius) {
        row[n_y * max_num_neighbors + count] = n_y;
        col[n_y * max_num_neighbors + count] = n_x;
        count++;
      }

      if (count >= max_num_neighbors) {
        break;
      }
    }
  }
}

torch::Tensor radius_cuda(torch::Tensor x, torch::Tensor y, torch::Tensor ptr_x,
                          torch::Tensor ptr_y, double r,
                          int64_t max_num_neighbors) {
  CHECK_CUDA(x);
  CHECK_CUDA(y);
  CHECK_CUDA(ptr_x);
  CHECK_CUDA(ptr_y);
  hipSetDevice(x.get_device());

  x = x.view({x.size(0), -1}).contiguous();
  y = y.view({y.size(0), -1}).contiguous();

  auto row = torch::full(y.size(0) * max_num_neighbors, -1, ptr_y.options());
  auto col = torch::full(y.size(0) * max_num_neighbors, -1, ptr_y.options());

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "radius_kernel", [&] {
    radius_kernel<scalar_t><<<ptr_x.size(0) - 1, THREADS, 0, stream>>>(
        x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(),
        ptr_x.data_ptr<int64_t>(), ptr_y.data_ptr<int64_t>(),
        row.data_ptr<int64_t>(), col.data_ptr<int64_t>(), r, max_num_neighbors,
        x.size(1));
  });

  auto mask = row != -1;
  return torch::stack({row.masked_select(mask), col.masked_select(mask)}, 0);
}
